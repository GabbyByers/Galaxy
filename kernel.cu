#include "hip/hip_runtime.h"
#include "kernel.cuh"

__global__ void kernel(int* a, int* b, int* c) {
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

void go(int* a, int* b, int* c, int arraySize) {
    hipSetDevice(0);

    int* dev_a = nullptr;
    int* dev_b = nullptr;
    int* dev_c = nullptr;
    hipMalloc((void**)&dev_a, arraySize * sizeof(int));
    hipMalloc((void**)&dev_b, arraySize * sizeof(int));
    hipMalloc((void**)&dev_c, arraySize * sizeof(int));

    hipMemcpy(dev_a, a, arraySize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, arraySize * sizeof(int), hipMemcpyHostToDevice);

    kernel <<<1, arraySize>>> (dev_a, dev_b, dev_c);
    hipDeviceSynchronize();

    hipMemcpy(c, dev_c, arraySize * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);
}